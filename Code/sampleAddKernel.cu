
#include <hip/hip_runtime.h>
#include <math.h>
#include <stdio.h>


__global__ void sampleAdd(int const * const A, int *B, int m) {

  
  //===========Initialize Variables====================
  
  int Bins=255; // How many cells your Histogram has (It is Image so hase 255)
  
  int L_Hist[256]; // TODO: Change the length of this image with Variable
  for (int PP=0;PP<=255;PP++){
	L_Hist[PP]=0; // Initialize the cells with the number 0
  }
  
  
  
  int cellsEveryThread=m/blockDim.x; // How much of the starter Array A Every thread will work with 
  
  
  int i = cellsEveryThread*threadIdx.x; // Where in the Array A every thread will start working
  
  
  //===========Initialize Variables END====================
  
  
  
  // ======================Create the Histogram==================
  int Posit;
  for (int Count=0;Count<cellsEveryThread;Count++)// Read all the 
  {
      if (i+Count<m)
	  {
		  Posit = A[i+Count];
		  L_Hist[Posit]=L_Hist[Posit]+1; // We don't need Atomic ADD because every 
		  
	  }
  }
  __syncthreads();
  
  
  
  __shared__ int L_PositionArray[8]; // Aftos o pinakas prepi na mirazete anamesa sta threads kai dimiourgite mesa apo ena bin olon ton threads
  
  int DD;
  for (int ki=0;ki<Bins;ki++){ // ki<Bins
	
	DD=L_Hist[ki];
	
	L_PositionArray[threadIdx.x]=DD;// Perno kathe fora ena simio apo ola ta Histogram olon ton threads
    
	__syncthreads();
	
	//==================Sum the Numbers and Right the Final Result to the table===============
	
	int tid=threadIdx.x;
	
	for (int s=blockDim.x/2;s>0;s>>=1){
	
	
		if (tid<s){
			
			int kks=tid+s;
			
			L_PositionArray[tid]=L_PositionArray[tid]+L_PositionArray[kks];
		}
		
		__syncthreads();
	}
	
	
	B[ki]=L_PositionArray[0];
	
	
	//==================Sum the Numbers and Right the Final Result to the table===============
	
	
	__syncthreads();
	
  }
  
  
 
}
  




















